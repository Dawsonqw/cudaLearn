#include <cstdio>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define THREADS_PER_BLOCK 256

int main(){
    const int N=32*1024*1024;
    float *input=(float*)malloc(N*sizeof(float));
    float *d_input;
    hipMalloc((void**)&d_input,N*sizeof(float));

    int  block_num=N / THREADS_PER_BLOCK;

    float *output=(float *)malloc(N/THREADS_PER_BLOCK*sizeof(float));
    float *d_output;
    hipMalloc((void**)&d_output,N/THREADS_PER_BLOCK*sizeof(float));

    float* result=(float*)malloc(N/THREADS_PER_BLOCK*sizeof(float));

    for(int i=0;i<N;i++){
        input[i]=2.0*(float)drand48()-1.0;
    }

    for(int i=0;i<block_num;i++){
        float sum=0.0;
        for(int j=0;j<THREADS_PER_BLOCK;j++){
            sum+=input[i*THREADS_PER_BLOCK+j];
        }
        result[i]=sum;
    }

    hipMemcpy(d_input,input,N*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid(block_num,1,1);
    dim3 Block(THREADS_PER_BLOCK,1,1);

    // TODO

    free(input);
    free(output);
    free(result);
    hipFree(d_input);
    hipFree(d_output);
    hipDeviceReset();
    printf("success\n");
    return 0;
}